﻿#include <GL/_Window.h>
#include <GL/_Texture.h>
#include <CUDA/_CUDA.h>
#include <_Math.h>
#include <_Time.h>
#include <RayTracing/_RayTracing.h>
#include <_STL.h>
#include <_BMP.h>
#include <random>

namespace OpenGL
{
	struct RayTrace :OpenGL
	{
		struct TextureData :Texture::Data
		{
			unsigned int size;
			float* data;


			TextureData(unsigned int _size, String<char> const& _data)
				:
				size(_size)
			{
				//_data.printInfo();
				data = (float*)malloc(size * sizeof(float));
				unsigned int n(0);
				for (unsigned int c0(0); c0 < size; ++c0)
				{
					unsigned int dn(0);
					sscanf(_data.data + n, "%f%n", data + c0, &dn);
					//printf("%.2f ", data[c0]);
					n += dn;
				}
			}
			~TextureData()
			{
				if (data)
				{
					free(data);
					data = nullptr;
				}
			}

			virtual void* pointer()override
			{
				return data;
			}
		};

		struct Renderer :Program
		{
			RayTracing::View view;
			Buffer viewBuffer;
			BufferConfig viewArray;
			VertexAttrib position;

			Renderer(SourceManager* _sm)
				:
				Program(_sm, "Renderer", Vector<VertexAttrib*>{&position}),
				view(),
				viewBuffer(&view),
				viewArray(&viewBuffer, ArrayBuffer),
				position(&viewArray, 0, VertexAttrib::two,
					VertexAttrib::Float, false, sizeof(Math::vec2<float>), 0, 0)
			{
				init();
			}
			virtual void initBufferData()override
			{
			}
			virtual void run()override
			{
				glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
				glClear(GL_COLOR_BUFFER_BIT);
				glDrawArrays(GL_TRIANGLE_FAN, 0, 4);
			}
		};

		SourceManager sm;
		bool sizeChanged;

		RayTracing::FrameScale frameScale;
		RayTracing::Transform transform;

		Buffer frameSizeBuffer;
		Buffer transBuffer;

		BufferConfig frameSizeUniform;
		BufferConfig transUniform;

		BMPCubeData cubeData;
		TextureCube cube;

		static constexpr unsigned int textureWidth = 64;
		TextureData textureDataCase1;
		TextureData textureDataCase2;
		TextureData textureDataAlphaM;
		Texture textureCase1;
		Texture textureCase2;
		Texture textureAlphaM;
		TextureConfig<TextureStorage2D>textureConfigCase1;
		TextureConfig<TextureStorage2D>textureConfigCase2;
		TextureConfig<TextureStorage1D>textureConfigCase2AlphaM;

		Renderer renderer;

		RayTrace(Math::vec2<unsigned int> const& _size)
			:
			sm(),
			sizeChanged(true),
			frameScale(),
			transform({ {60.0},{0.002,0.9,0.001},{0.02},{0,0,20},700.0 }),
			frameSizeBuffer(&frameScale),
			transBuffer(&transform.bufferData),
			frameSizeUniform(&frameSizeBuffer, UniformBuffer, 0),
			transUniform(&transBuffer, UniformBuffer, 1),
			cubeData("resources/room/"),
			cube(&cubeData, 0, RGBA32f, 1, cubeData.bmp[0].header.width, cubeData.bmp[0].header.height),
			textureDataCase1(textureWidth* textureWidth, sm.folder.find("resources/unified.txt").readText()),
			textureDataCase2(textureWidth* textureWidth, sm.folder.find("resources/case2.txt").readText()),
			textureDataAlphaM(textureWidth, sm.folder.find("resources/alpha_m.txt").readText()),
			textureCase1(&textureDataCase1, 1),
			textureCase2(&textureDataCase2, 2),
			textureAlphaM(&textureDataAlphaM, 3),
			textureConfigCase1(&textureCase1, Texture2D, R32f, 1, textureWidth, textureWidth),
			textureConfigCase2(&textureCase2, Texture2D, R32f, 1, textureWidth, textureWidth),
			textureConfigCase2AlphaM(&textureAlphaM, Texture1D, R32f, 1, textureWidth),
			renderer(&sm)
		{
			cube.dataInit(0, TextureInputBGRInt, TextureInputUByte);
			textureConfigCase1.dataInit(0, TextureInputR, TextureInputFloat);
			textureConfigCase2.dataInit(0, TextureInputR, TextureInputFloat);
			textureConfigCase2AlphaM.dataInit(0, TextureInputR, TextureInputFloat);

			renderer.use();
			cube.bindUnit();
			textureCase1.bindUnit();
			textureCase2.bindUnit();
			textureAlphaM.bindUnit();
			
			using namespace TextureParameter;
			cube.parameteri(TextureMinFilter, MinFilter_Linear);
			textureConfigCase1.parameteri(TextureMinFilter, MinFilter_Nearest);
			textureConfigCase2.parameteri(TextureMinFilter, MinFilter_Linear);
			textureConfigCase2AlphaM.parameteri(TextureMinFilter, MinFilter_Nearest);

			//textureConfigCase2.parameteri(TextureWarpS, Wrap_ClampToEdge);
			//textureConfigCase2.parameteri(TextureWarpT, Wrap_ClampToEdge);

			float black[4]{ 0 };

			//glTexParameterfv(GL_TEXTURE_2D, GL_TEXTURE_BORDER_COLOR, black);

			glEnable(GL_TEXTURE_CUBE_MAP_SEAMLESS);
		}
		virtual void init(FrameScale const& _size) override
		{
			glViewport(0, 0, _size.w, _size.h);
			transform.init(_size);
			frameScale.scale = { unsigned int(_size.w),unsigned int(_size.h) };
			renderer.viewArray.dataInit();
			frameSizeUniform.dataInit();
			sizeChanged = false;
			transUniform.dataInit();
		}
		virtual void run() override
		{
			transform.bufferData.trans.times = 0;
			if (sizeChanged)
			{
				glViewport(0, 0, frameScale.scale.data[0], frameScale.scale.data[1]);
				renderer.use();
				frameSizeUniform.refreshData();
				transform.bufferData.trans.times = 0;
				sizeChanged = false;
			}
			transform.operate();
			if (transform.updated)
			{
				transUniform.refreshData();
				transform.updated = false;
			}
			renderer.use();
			renderer.run();
		}
		virtual void frameSize(int _w, int _h) override
		{
			frameScale.scale = { unsigned int(_w),unsigned int(_h) };
			transform.persp.y = _h;
			transform.persp.updated = true;
			sizeChanged = true;
		}
		virtual void framePos(int, int) override
		{
		}
		virtual void frameFocus(int) override
		{
		}
		virtual void mouseButton(int _button, int _action, int _mods) override
		{
			switch (_button)
			{
			case GLFW_MOUSE_BUTTON_LEFT:transform.mouse.refreshButton(0, _action); break;
			case GLFW_MOUSE_BUTTON_MIDDLE:transform.mouse.refreshButton(1, _action); break;
			case GLFW_MOUSE_BUTTON_RIGHT:transform.mouse.refreshButton(2, _action); break;
			}
		}
		virtual void mousePos(double _x, double _y) override
		{
			transform.mouse.refreshPos(_x, _y);
		}
		virtual void mouseScroll(double _x, double _y) override
		{
			if (_y != 0.0)
				transform.scroll.refresh(_y);
		}
		virtual void key(GLFWwindow* _window, int _key, int _scancode, int _action, int _mods) override
		{
			switch (_key)
			{
			case GLFW_KEY_ESCAPE:
				if (_action == GLFW_PRESS)
					glfwSetWindowShouldClose(_window, true);
				break;
			case GLFW_KEY_A:transform.key.refresh(0, _action); break;
			case GLFW_KEY_D:transform.key.refresh(1, _action); break;
			case GLFW_KEY_W:transform.key.refresh(2, _action); break;
			case GLFW_KEY_S:transform.key.refresh(3, _action); break;
			}
		}
	};
}

int main()
{
	OpenGL::OpenGLInit init(4, 5);
	Window::Window::Data winPara
	{
		"BlackHoleRendering",
		{
			{1200, 1200},
			true, false,
		}
	};
	Window::WindowManager wm(winPara);
	OpenGL::RayTrace test({ 1200, 1200 });
	wm.init(0, &test);
	glfwSwapInterval(1);
	FPS fps;
	fps.refresh();
	while (!wm.close())
	{
		wm.pullEvents();
		wm.render();
		wm.swapBuffers();
		fps.refresh();
		fps.printFPS(1);
	}
	return 0;
}
