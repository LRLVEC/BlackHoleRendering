﻿#include <GL/_Window.h>
#include <GL/_Texture.h>
#include <CUDA/_CUDA.h>
#include <_Math.h>
#include <_Time.h>
#include <RayTracing/_RayTracing.h>
#include <_STL.h>
#include <_BMP.h>
#include <random>

namespace OpenGL
{
	struct RayTrace :OpenGL
	{
		struct Renderer :Program
		{
			RayTracing::View view;
			Buffer viewBuffer;
			BufferConfig viewArray;
			VertexAttrib position;

			Renderer(SourceManager* _sm)
				:
				Program(_sm, "Renderer", Vector<VertexAttrib*>{&position}),
				view(),
				viewBuffer(&view),
				viewArray(&viewBuffer, ArrayBuffer),
				position(&viewArray, 0, VertexAttrib::two,
					VertexAttrib::Float, false, sizeof(Math::vec2<float>), 0, 0)
			{
				init();
			}
			virtual void initBufferData()override
			{
			}
			virtual void run()override
			{
				glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
				glClear(GL_COLOR_BUFFER_BIT);
				glDrawArrays(GL_TRIANGLE_FAN, 0, 4);
			}
		};

		SourceManager sm;
		bool sizeChanged;
		RayTracing::FrameScale frameScale;
		RayTracing::Transform transform;
		Buffer frameSizeBuffer;
		Buffer transBuffer;
		BufferConfig frameSizeUniform;
		BufferConfig transUniform;
		BMPCubeData cubeData;
		TextureCube cube;
		Renderer renderer;

		RayTrace(Math::vec2<unsigned int> const& _size)
			:
			sm(),
			sizeChanged(true),
			frameScale(),
			transform({ {60.0},{0.002,0.9,0.001},{0.01},{0,0,0},700.0 }),
			frameSizeBuffer(&frameScale),
			transBuffer(&transform.bufferData),
			frameSizeUniform(&frameSizeBuffer, UniformBuffer, 0),
			transUniform(&transBuffer, UniformBuffer, 1),
			cubeData("resources/room/"),
			cube(&cubeData, 2, RGBA32f, 1, cubeData.bmp[0].header.width, cubeData.bmp[0].header.height),
			renderer(&sm)
		{
			cube.dataInit(0, TextureInputBGRInt, TextureInputUByte);
			renderer.use();
			cube.bindUnit();
			glEnable(GL_TEXTURE_CUBE_MAP_SEAMLESS);

		}
		virtual void init(FrameScale const& _size) override
		{
			glViewport(0, 0, _size.w, _size.h);
			transform.init(_size);
			frameScale.scale = { unsigned int(_size.w),unsigned int(_size.h) };
			renderer.viewArray.dataInit();
			frameSizeUniform.dataInit();
			sizeChanged = false;
			transUniform.dataInit();
		}
		virtual void run() override
		{
			transform.bufferData.trans.times = 0;
			if (sizeChanged)
			{
				glViewport(0, 0, frameScale.scale.data[0], frameScale.scale.data[1]);
				renderer.use();
				frameSizeUniform.refreshData();
				transform.bufferData.trans.times = 0;
				sizeChanged = false;
			}
			transform.operate();
			if (transform.updated)
			{
				transUniform.refreshData();
				transform.updated = false;
			}
			renderer.use();
			renderer.run();
		}
		virtual void frameSize(int _w, int _h) override
		{
			frameScale.scale = { unsigned int(_w),unsigned int(_h) };
			transform.persp.y = _h;
			transform.persp.updated = true;
			sizeChanged = true;
		}
		virtual void framePos(int, int) override
		{
		}
		virtual void frameFocus(int) override
		{
		}
		virtual void mouseButton(int _button, int _action, int _mods) override
		{
			switch (_button)
			{
			case GLFW_MOUSE_BUTTON_LEFT:transform.mouse.refreshButton(0, _action); break;
			case GLFW_MOUSE_BUTTON_MIDDLE:transform.mouse.refreshButton(1, _action); break;
			case GLFW_MOUSE_BUTTON_RIGHT:transform.mouse.refreshButton(2, _action); break;
			}
		}
		virtual void mousePos(double _x, double _y) override
		{
			transform.mouse.refreshPos(_x, _y);
		}
		virtual void mouseScroll(double _x, double _y) override
		{
			if (_y != 0.0)
				transform.scroll.refresh(_y);
		}
		virtual void key(GLFWwindow* _window, int _key, int _scancode, int _action, int _mods) override
		{
			switch (_key)
			{
			case GLFW_KEY_ESCAPE:
				if (_action == GLFW_PRESS)
					glfwSetWindowShouldClose(_window, true);
				break;
			case GLFW_KEY_A:transform.key.refresh(0, _action); break;
			case GLFW_KEY_D:transform.key.refresh(1, _action); break;
			case GLFW_KEY_W:transform.key.refresh(2, _action); break;
			case GLFW_KEY_S:transform.key.refresh(3, _action); break;
			}
		}
	};
}

int main()
{
	OpenGL::OpenGLInit init(4, 5);
	Window::Window::Data winPara
	{
		"BlackHoleRendering",
		{
			{1920,1080},
			true, false,
		}
	};
	Window::WindowManager wm(winPara);
	OpenGL::RayTrace test({ 1920,1080 });
	wm.init(0, &test);
	glfwSwapInterval(0);
	FPS fps;
	fps.refresh();
	while (!wm.close())
	{
		wm.pullEvents();
		wm.render();
		wm.swapBuffers();
		fps.refresh();
		fps.printFPS(1);
	}
	return 0;
}
